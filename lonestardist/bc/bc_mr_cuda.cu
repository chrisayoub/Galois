#include "hip/hip_runtime.h"
#include "bc_mr_cuda.cuh"

__global__
void InitializeIteration(
		CSRGraph graph,
		unsigned int __begin, unsigned int __end,
		uint32_t* p_roundIndexToSend,
		CUDATree* p_dTree,
		BCData** p_sourceData,
		uint64_t* nodesToConsider, unsigned numSourcesPerRound)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  for (index_type src = __begin + tid; src < __end; src += nthreads)
  {
	  p_roundIndexToSend[src] = infinity;
	  CUDATree dTree = p_dTree[src];
	  dTree.initialize();

	  BCData* bcArray = p_sourceData[src];
	  // Loop through sources
	  for (unsigned i = 0; i < numSourcesPerRound; i++) {
		  if (nodesToConsider[i] == graph.node_data[src]) {
			  // This is a source node
			  bcArray[i].minDistance = 0;
			  bcArray[i].shortPathCount = 1;
			  bcArray[i].dependencyValue = 0.0;
			  dTree.setDistance(i, 0);
		  } else {
			  // This is a non-source node
			  bcArray[i].minDistance = infinity;
			  bcArray[i].shortPathCount = 0;
			  bcArray[i].dependencyValue = 0.0;
		  }
	  }
  }
}

__global__
void FindMessageToSync(CSRGraph graph,
		unsigned int __begin, unsigned int __end,
		HGAccumulator<uint32_t> dga,
		const uint32_t roundNumber,
		uint32_t* p_roundIndexToSend,
		CUDATree* p_dTree,
		BCData** p_sourceData) {

	unsigned tid = TID_1D;
	unsigned nthreads = TOTAL_THREADS_1D;

	__shared__ hipcub::BlockReduce<uint32_t, TB_SIZE>::TempStorage dga_ts;
	dga.thread_entry();

	for (index_type src = __begin + tid; src < __end; src += nthreads)
	{
		BCData* sourceData = p_sourceData[src];
		uint32_t* roundIndexToSend = &p_roundIndexToSend[src];
		CUDATree dTree = p_dTree[src];

		uint32_t newRoundIndex = dTree.getIndexToSend(roundNumber);
		*roundIndexToSend = newRoundIndex;

		if (newRoundIndex != infinity) {
			if (sourceData[newRoundIndex].minDistance != 0) {
				// TODO pass this as param so can set
//				bitset_minDistances.set(curNode);
			}
			dga.reduce(1);
		} else if (dTree.moreWork()) {
			dga.reduce(1);
		}
	}

	dga.thread_exit<hipcub::BlockReduce<uint32_t, TB_SIZE>>(dga_ts);
}

void InitializeGraph_allNodes_cuda(struct CUDA_Context* ctx, unsigned int vectorSize)
{
  // Init sourceData array to new array of size vectorSize
  // Number of nodes * array size for each node
  ctx->sourceData.data = Shared<BCData*>((size_t) (ctx->gg.nnodes * vectorSize));

  // Set all memory to 0
  reset_CUDA_context(ctx);

  // Finish op
  hipDeviceSynchronize();
  check_cuda_kernel;
}

void InitializeIteration_allNodes_cuda(struct CUDA_Context* ctx,
		const std::vector<uint64_t>& nodesToConsider,
		unsigned numSourcesPerRound) {
	// Copy source array to GPU
	uint64_t* nodesArr;
	size_t arrSize = nodesToConsider.size() * sizeof(uint64_t);
	hipMalloc(&nodesArr, arrSize);
	hipMemcpy(nodesArr, nodesToConsider.data(), arrSize, hipMemcpyHostToDevice);

	// Sizing
	dim3 blocks;
	dim3 threads;
	kernel_sizing(blocks, threads);

	// Kernel call
	InitializeIteration <<<blocks, threads>>>(ctx->gg, 0, ctx->gg.nnodes,
			ctx->roundIndexToSend.data.gpu_wr_ptr(),
			ctx->dTree.data.gpu_wr_ptr(),
			ctx->sourceData.data.gpu_wr_ptr(),
			nodesArr, numSourcesPerRound);

	// Clean up
	hipFree(nodesArr);
	hipDeviceSynchronize();
	check_cuda_kernel;
}

void FindMessageToSync_allNodes_cuda(struct CUDA_Context* ctx, const uint32_t roundNumber, uint32_t & dga) {
	// Sizing
	dim3 blocks;
	dim3 threads;
	kernel_sizing(blocks, threads);

	// Accumulator
	HGAccumulator<uint32_t> _dga;
	Shared<uint32_t> dgaval  = Shared<uint32_t>(1);
	*(dgaval.cpu_wr_ptr()) = 0;
	_dga.rv = dgaval.gpu_wr_ptr();

	// Kernel call
	FindMessageToSync <<<blocks, threads>>>(
			ctx->gg, 0, ctx->gg.nnodes,
			_dga, roundNumber,
			ctx->roundIndexToSend.data.gpu_wr_ptr(),
			ctx->dTree.data.gpu_wr_ptr(),
			ctx->sourceData.data.gpu_wr_ptr());

	// Clean up
	hipDeviceSynchronize();
	check_cuda_kernel;

	// Copy back return value
	dga = *(dgaval.cpu_rd_ptr());
}
