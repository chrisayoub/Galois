#include "hip/hip_runtime.h"
#include "bc_mr_cuda.cuh"

__global__
void InitializeIteration(
		CSRGraph graph,
		unsigned int __begin, unsigned int __end,
		uint32_t* p_roundIndexToSend,
		CUDATree* p_dTree,
		uint32_t** p_minDistance,
		ShortPathType** p_shortPathCount,
		float** p_dependencyValue,
		uint64_t* nodesToConsider, unsigned numSourcesPerRound)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  for (index_type src = __begin + tid; src < __end; src += nthreads)
  {
	  p_roundIndexToSend[src] = infinity;
	  CUDATree dTree = p_dTree[src];
	  dTree.initialize();

	  uint32_t* minDistances = p_minDistance[src];
	  ShortPathType* shortPathCounts = p_shortPathCount[src];
	  float* dependencyValues = p_dependencyValue[src];
	  // Loop through sources
	  for (unsigned i = 0; i < numSourcesPerRound; i++) {
		  if (nodesToConsider[i] == graph.node_data[src]) {
			  // This is a source node
			  minDistances[i] = 0;
			  shortPathCounts[i] = 1;
			  dependencyValues[i] = 0.0;
			  dTree.setDistance(i, 0);
		  } else {
			  // This is a non-source node
			  minDistances[i] = infinity;
			  shortPathCounts[i] = 0;
			  dependencyValues[i] = 0.0;
		  }
	  }
  }
}

// TOOD update for new GPU data layout
/*
__global__
void FindMessageToSync(CSRGraph graph,
		unsigned int __begin, unsigned int __end,
		HGAccumulator<uint32_t> dga,
		const uint32_t roundNumber,
		uint32_t* p_roundIndexToSend,
		CUDATree* p_dTree,
		BCData** p_sourceData) {

	unsigned tid = TID_1D;
	unsigned nthreads = TOTAL_THREADS_1D;

	__shared__ hipcub::BlockReduce<uint32_t, TB_SIZE>::TempStorage dga_ts;
	dga.thread_entry();

	for (index_type src = __begin + tid; src < __end; src += nthreads)
	{
		BCData* sourceData = p_sourceData[src];
		uint32_t* roundIndexToSend = &p_roundIndexToSend[src];
		CUDATree dTree = p_dTree[src];

		uint32_t newRoundIndex = dTree.getIndexToSend(roundNumber);
		*roundIndexToSend = newRoundIndex;

		if (newRoundIndex != infinity) {
			if (sourceData[newRoundIndex].minDistance != 0) {
				// TODO pass this as param so can set
//				bitset_minDistances.set(curNode);
			}
			dga.reduce(1);
		} else if (dTree.moreWork()) {
			dga.reduce(1);
		}
	}

	dga.thread_exit<hipcub::BlockReduce<uint32_t, TB_SIZE>>(dga_ts);
}
*/

void InitializeGraph_allNodes_cuda(struct CUDA_Context* ctx, unsigned int vectorSize)
{
	// Init arrays to be to new arrays of size vectorSize
	// Number of nodes * array size for each node
	size_t arraySize = (size_t) (ctx->gg.nnodes * vectorSize);
	ctx->minDistance.data = Shared<uint32_t*>(arraySize);
	ctx->shortPathCount.data = Shared<ShortPathType*>(arraySize);
	ctx->dependencyValue.data = Shared<float*>(arraySize);

	// Set all memory to 0
	reset_CUDA_context(ctx);

	// Finish op
	hipDeviceSynchronize();
	check_cuda_kernel;
}

void InitializeIteration_allNodes_cuda(struct CUDA_Context* ctx,
		const std::vector<uint64_t>& nodesToConsider,
		unsigned numSourcesPerRound) {
	// Copy source array to GPU
	uint64_t* nodesArr;
	size_t arrSize = nodesToConsider.size() * sizeof(uint64_t);
	hipMalloc(&nodesArr, arrSize);
	hipMemcpy(nodesArr, nodesToConsider.data(), arrSize, hipMemcpyHostToDevice);

	// Sizing
	dim3 blocks;
	dim3 threads;
	kernel_sizing(blocks, threads);

	// Kernel call
	InitializeIteration <<<blocks, threads>>>(ctx->gg, 0, ctx->gg.nnodes,
			ctx->roundIndexToSend.data.gpu_wr_ptr(),
			ctx->dTree.data.gpu_wr_ptr(),
			ctx->minDistance.data.gpu_wr_ptr(),
			ctx->shortPathCount.data.gpu_wr_ptr(),
			ctx->dependencyValue.data.gpu_wr_ptr(),
			nodesArr, numSourcesPerRound);

	// Clean up
	hipFree(nodesArr);
	hipDeviceSynchronize();
	check_cuda_kernel;
}

/*  TOOD re-enable and update
void FindMessageToSync_allNodes_cuda(struct CUDA_Context* ctx, const uint32_t roundNumber, uint32_t & dga) {
	// Sizing
	dim3 blocks;
	dim3 threads;
	kernel_sizing(blocks, threads);

	// Accumulator
	HGAccumulator<uint32_t> _dga;
	Shared<uint32_t> dgaval  = Shared<uint32_t>(1);
	*(dgaval.cpu_wr_ptr()) = 0;
	_dga.rv = dgaval.gpu_wr_ptr();

	// Kernel call
	FindMessageToSync <<<blocks, threads>>>(
			ctx->gg, 0, ctx->gg.nnodes,
			_dga, roundNumber,
			ctx->roundIndexToSend.data.gpu_wr_ptr(),
			ctx->dTree.data.gpu_wr_ptr(),
			ctx->sourceData.data.gpu_wr_ptr());

	// Clean up
	hipDeviceSynchronize();
	check_cuda_kernel;

	// Copy back return value
	dga = *(dgaval.cpu_rd_ptr());
}
*/
